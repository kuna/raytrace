#include "hip/hip_runtime.h"
#include <iostream>
#include "pnglib.h"

__global__ void VecAdd(float* A, float* B, float* C) {
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

//
// void setRGB
//
// sets RGB pixel data on png_byte
//
typedef unsigned short int uint8_t;
#define getR(v) ((v) & 0xFF)
#define getG(v) (((v) >> 2) & 0xFF)
#define getB(v) (((v) >> 4) & 0xFF)
#define getA(v) (((v) >> 6) & 0xFF)
inline void setRGB(int *ptr, uint8_t R, uint8_t G, uint8_t B)
{
	*ptr = R | (G << 2) | (B << 4);
}



int main () {
	float* A_t, *B_t, *C_t;
	hipMalloc(&A_t, sizeof(float)*100);
	hipMalloc(&B_t, sizeof(float)*100);
	hipMalloc(&C_t, sizeof(float)*100);

	// test work
	VecAdd<<<1, 100 >>> (A_t, B_t, C_t);

	hipFree(A_t);
	hipFree(B_t);
	hipFree(C_t);

	// test writing image
	int test[64*64];
	memset(test, 0xFF, sizeof(int)*64*64);
	setRGB(&test[0], 255, 0, 0);
	writeImage("test.png", 64, 64, test, "test");

	// test reading image
	int width=0, height=0;
	readImage("kaede.png", &width, &height, test);
	std::cout << "image info: " << width << ", " <<  height << ", "
		<< getR(test[0]) << ", " << getG(test[0]) << ", " << getB(test[0]);

	return 0;
}
